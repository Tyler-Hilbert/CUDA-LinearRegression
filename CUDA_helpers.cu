//%%writefile CUDA_helpers.cu

// CUDA and cuBLAS error checkers

#ifndef __CUDA_HELPERS__
#define __CUDA_HELPERS__

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>


#define CUDA_CHECK(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) 
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

// Macro for cuBLAS error checking
#define CUBLAS_CHECK(ans) { \
    if ((ans) != HIPBLAS_STATUS_SUCCESS) { \
        fprintf(stderr, "cuBLAS Error: %s %s %d\n", cublasGetErrorString(ans), __FILE__, __LINE__); \
        exit(ans); \
    } \
}

// Inline function to map cublasStatus_t to string within the macro
inline const char* cublasGetErrorString(hipblasStatus_t status) {
    switch (status) {
        case HIPBLAS_STATUS_SUCCESS:          return "CUBLAS_STATUS_SUCCESS";
        case HIPBLAS_STATUS_NOT_INITIALIZED:  return "CUBLAS_STATUS_NOT_INITIALIZED";
        case HIPBLAS_STATUS_ALLOC_FAILED:     return "CUBLAS_STATUS_ALLOC_FAILED";
        case HIPBLAS_STATUS_INVALID_VALUE:    return "CUBLAS_STATUS_INVALID_VALUE";
        case HIPBLAS_STATUS_ARCH_MISMATCH:    return "CUBLAS_STATUS_ARCH_MISMATCH";
        case HIPBLAS_STATUS_MAPPING_ERROR:    return "CUBLAS_STATUS_MAPPING_ERROR";
        case HIPBLAS_STATUS_EXECUTION_FAILED: return "CUBLAS_STATUS_EXECUTION_FAILED";
        case HIPBLAS_STATUS_INTERNAL_ERROR:   return "CUBLAS_STATUS_INTERNAL_ERROR";
        case HIPBLAS_STATUS_NOT_SUPPORTED:    return "CUBLAS_STATUS_NOT_SUPPORTED";
        case HIPBLAS_STATUS_UNKNOWN:    return "CUBLAS_STATUS_LICENSE_ERROR";
        default:                             return "UNKNOWN_CUBLAS_STATUS";
    }
}

#endif // __CUDA_HELPERS__